#include "hip/hip_runtime.h"
#include "constant.cuh"
#include "hip/hip_runtime.h"

#define DOUBLE float

// now need to solve problem with 2d array. 

// suppelmentary functions that need to be solved first:
// this is optimazation. Will do after testing all the main calculation in solver is done
// Tinh KhoUot
// Find Calculation Limit
// Htuongdoi
// GiatriHtaiZ
// VTHZ()
// K_factor()

__device__ int calculate_index(int M){
    int thrx = blockIdx.x * blockDim.x + threadIdx.x;
    int thry = blockIdx.y * blockDim.y + threadIdx.y;
    return thrx * (blockDim.y * gridDim.y) + thry;
}

//-----------------------------------Reynold Eq Solver--------------------------------------------
__device__ void tridiag(int sn, DOUBLE* AA, DOUBLE* BB, DOUBLE* CC, DOUBLE*DD, DOUBLE *x, 
    DOUBLE *Ap, DOUBLE *Bp, DOUBLE *ep){
    Ap[1] = - CC[1] / BB[1];
    Bp[1] = DD[1] / BB[1];
    //printf("hello from Thomas, sn = % d\n", sn);
    for (int i = 2; i < sn; i++){
        //printf("i = %d AA: %f , BB: %f , CC: %f , DD: % f\n", i, AA[i], BB[i], CC[i], DD[i] );
        ep[i] = AA[i] * Ap[i - 1] + BB[i];
        Ap[i] = -CC[i] / ep[i];
        Bp[i] = (DD[i] - (AA[i] * Bp[i - 1])) / ep[i];
    }

    x[sn] = (DD[sn] - (AA[sn] * Bp[sn - 1])) / (BB[sn] + (AA[sn] * Ap[sn - 1]));

    for (int i = sn - 1; i > 0; i--){
        x[i] = Bp[i] + (Ap[i] * x[i + 1]);
        //printf("thread: %d , i: %d x: %.15f \n", (blockIdx.x*blockDim.x + threadIdx.x) + 3, i, x[i]);
    }
}

__device__ void bienrandau(int first, int last, DOUBLE* AA, DOUBLE* BB, DOUBLE* CC, DOUBLE*DD,
    DOUBLE *a1, DOUBLE *b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2){

    //printf("bienrandau is called \n");
    for (int i = first; i <= last; i++){
        AA[(i - first) * 2 + 1] = a2[i];
        BB[(i - first) * 2 + 1] = 1;
        CC[(i - first) * 2 + 1] = c2[i];
        DD[(i - first) * 2 + 1] = d2[i];
     
        AA[(i - first + 1) * 2] = a1[i];
        BB[(i - first + 1) * 2] = b1[i];
        CC[(i - first + 1) * 2] = c1[i];
        DD[(i - first + 1) * 2] = d1[i];
    }
}


__device__ void bienlongdau(int first, int last, DOUBLE* AA, DOUBLE* BB, DOUBLE* CC, DOUBLE*DD,
    DOUBLE *a1, DOUBLE *b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2){

    //printf("bienlongdau is called \nFirst = %d , last = %d\n", first, last);
    for (int i = first; i < last; i++){
        AA[(i - first) * 2 + 1] = a1[i];
        BB[(i - first) * 2 + 1] = b1[i];  
        CC[(i - first) * 2 + 1] = c1[i];
        DD[(i - first) * 2 + 1] = d1[i];

        AA[(i - first + 1) * 2] = a2[i + 1];
        BB[(i - first + 1) * 2] = 1;
        CC[(i - first + 1) * 2] = c2[i + 1];
        DD[(i - first + 1) * 2] = d2[i + 1];
    }

}

__device__ void update_abcd_at_l(int first, int last, bool bienran1, bool bienran2, DOUBLE *a1, DOUBLE *b1, DOUBLE *c1, DOUBLE *d1,
    DOUBLE* f1, DOUBLE* f2, DOUBLE* f3, DOUBLE f4, DOUBLE* f5, DOUBLE* f6, DOUBLE* f7, DOUBLE* f8){
    if (last - first > 1){
        if (bienran1){
            a1[first] = f4;
            b1[first] = f2[first] - (f3[first] * f7[first+ 1] / f6[first+ 1]);
            c1[first] = - f4 - (f3[first] / f6[first+ 1]);
            d1[first] = f5[first] - (f3[first] * f8[first+ 1] / f6[first+ 1]);
        }
            
        else{
            a1[first] = f4 - f1[first] / f7[first];
            b1[first] = f2[first] - (f1[first] * f6[first] / f7[first]) - (f3[first] * f7[first+ 1] / f6[first+ 1]);
            c1[first] = -f4 - (f3[first] / f6[first+ 1]);
            d1[first] = f5[first] - (f1[first] * f8[first] / f7[first]) - (f3[first] * f8[first+ 1] / f6[first+ 1]);
        }
        if (last - first > 2){
            for(int i = first + 1; i < last - 1; i++){
                a1[i] = f4 - f1[i] / f7[i];
                b1[i] = f2[i] - (f1[i] * f6[i] / f7[i]) - (f3[i] * f7[i + 1] / f6[i + 1]);
                c1[i] = -f4 - (f3[i] / f6[i + 1]);
                d1[i] = f5[i] - (f1[i] * f8[i] / f7[i]) - (f3[i] * f8[i + 1] / f6[i + 1]);
            }
        }

        if (bienran2){
            a1[last - 1] = f4 - f1[last -1] / f7[last -1];
            b1[last - 1] = f2[last - 1] - (f1[last - 1] * f6[last -1] / f7[last - 1]);
            c1[last - 1] = - f4;
            d1[last - 1] = f5[last - 1] - (f1[last - 1] * f8[last - 1] / f7[last - 1]);
        }
        else{
            a1[last - 1] = f4 - f1[last - 1] / f7[last - 1];
            b1[last - 1] = f2[last - 1] - (f1[last - 1] * f6[last - 1] / f7[last - 1]) - (f3[last - 1] * f7[last] / f6[last]);
            c1[last - 1] = -f4 - (f3[last - 1] / f6[last]);
            d1[last - 1] = f5[last - 1] - (f1[last - 1] * f8[last - 1] / f7[last - 1]) - (f3[last - 1] * f8[last] / f6[last]);
        }
    }
    else{
        //printf("Overflow, comparision btw DOUBLE and int\n");
        a1[first] = f4;
        b1[first] = f2[first];
        d1[first] = f5[first];
        c1[first] = -f4;
    }
}



__device__ int boundary_config(bool isU, int first, int last, bool bienran1, 
    bool bienran2, DOUBLE ubp_or_vbt, DOUBLE ubt_or_vbd, DOUBLE TZ_r, DOUBLE TZ_l, bool dkBienQ_1, bool dkBienQ_2, int dkfr,
    DOUBLE *a1, DOUBLE *b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2, DOUBLE* AA, DOUBLE* BB, DOUBLE* CC, DOUBLE* DD){
    int sn;
    bool isBienran;
    if (bienran1){
        // attention
        //printf("gpu bien: %d, %d\n", dkBienQ_2, dkBienQ_1);
        bienrandau(first, last, AA, BB, CC, DD, a1, b1, c1, d1, a2, b2, c2, d2);
        DD[1] = d2[first];
        // ran - long
        if (bienran2 == false){
            if ((dkBienQ_2) && (last == dkfr)){         //r == dkfr:   // Kiem tra lai phan nay
                sn = 2 * (last - first) + 1;
                // attention 
                AA[sn] = a2[last];
                BB[sn] = 1;
                DD[sn] = d2[last] - c2[last] * ubp_or_vbt;
            }
            else{
                //printf("ran - long \n");
                sn = 2 * (last - first) ;
                AA[sn] = a1[last - 1];
                BB[sn] = b1[last - 1];
                DD[sn] = d1[last - 1] - c1[last - 1] * TZ_r;
            }
        }
        // ran - ran
        else{
            //printf("ran - ran \n");
            sn = 2 * (last - first) + 1;
            AA[sn] = a2[last];
            BB[sn] = 1;
            DD[sn] = d2[last];
        }
    }
    // long
    else{
        if ((dkBienQ_1) && (first == 2)){
            bienrandau(first, last, AA, BB, CC, DD, a1, b1, c1, d1, a2, b2, c2, d2);
            DD[1] = d2[first] - a2[first] * ubt_or_vbd;
            // thieu bb[1] va cc[1] cho truong hop vz, hoi lai co
            isBienran = true;
        }
        else{
            bienlongdau(first, last, AA, BB, CC, DD, a1, b1, c1, d1, a2, b2, c2, d2);
            BB[1] = b1[first];
            CC[1] = c1[first];
            DD[1] = d1[first] - a1[first] * TZ_l;
            isBienran = false;
        }
        // long - long
        if (bienran2 == false){ // variable isbienran is equivalent with variable text in original code
            if ((dkBienQ_1) && (last == dkfr)){     //r == dkfr: // BienQ[0] && r == M trong truong hop giaianv
                sn = 2 * (last - first);
                if (isBienran)
                    sn += 1;
                AA[sn] = a2[last];
                BB[sn] = 1;
                DD[sn] = d2[last] - c2[last] * ubp_or_vbt;
            }
            else{
                sn = 2 * (last - first);

                if (!isBienran)
                    sn -= 1;
                AA[sn] = a1[last - 1];
                BB[sn] = b1[last - 1];
                DD[sn] = d1[last - 1] - c1[last - 1] * TZ_r;
            }
        }
        else{
            sn = 2 * (last - first);
            if (isBienran)
                sn += 1;
            // AA[sn] = f7[last] 
            // this line is modified for the canal case
            AA[sn] = a2[last];
            BB[sn] = 1;
            DD[sn] = d2[last];
        }
    }
   
    return sn;

}

// missing: dXbp dT dTchia2dX g dYbp Windx() Tsxw DX2

// if change to  2d: need to change all parameter, 2d array fix element access
// if use 1d: need to change element access, which involve offset
__device__ void uzSolver(int offset, int N, int M, int first, int last, int jpos, bool bienran1, bool bienran2, 
    DOUBLE dXbp, DOUBLE dYbp, DOUBLE dT, DOUBLE dTchia2dX, DOUBLE dTchia2dY, DOUBLE* Tsxw, // constants
    DOUBLE* v, DOUBLE* u, DOUBLE* z, DOUBLE* Htdu, DOUBLE* Htdv, DOUBLE* VISCOIDX, DOUBLE* t_u, DOUBLE* t_z, 
    DOUBLE* ubt, DOUBLE* ubp, DOUBLE* H_moi, DOUBLE* Kx1, bool* bienQ,
    DOUBLE *f1, DOUBLE *f2, DOUBLE *f3, DOUBLE *f5, DOUBLE *f6, DOUBLE *f7, DOUBLE *f8,
    DOUBLE *a1, DOUBLE* b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2,
    DOUBLE *AA, DOUBLE *BB, DOUBLE *CC, DOUBLE *DD, DOUBLE *Ap, DOUBLE *Bp, DOUBLE* ep, DOUBLE* x){
    int j = jpos;
    DOUBLE f4 = 2 * g * dTchia2dX;
    DOUBLE p = 0.0;
    DOUBLE q = 0.0;
    // memory offset for 2d arrays:
    // 2d arrays in use: v, u, VISCOIDX, Htdu, Htdv, Kx1, Ky1, H_moi,  Tsxw, t_u, t_z 
    // same offset: u, v, Htdu, Htdv, t_u, t_z, Hmoi, Tsxw : N + 3
    // diffent offset: VISCOIDX, Kx1, Ky1, : MUST change to make these has same offset

    for (int i = first; i < last; i++){
        DOUBLE vtb = (v[i * offset +  j - 1] + v[i * offset + j] + v[(i + 1) * offset + j - 1] + v[(i + 1) * offset + j]) * 0.25;
        f1[i] = dTchia2dX * u[i * offset + j] + VISCOIDX[i * offset + j] * dT / dXbp;
        f2[i] = -(2 + Kx1[i * offset + j] * dT * sqrt(u[i * offset + j] * u[i * offset + j] + vtb * vtb) / Htdu[i * offset + j] + (2 * dT * VISCOIDX[i * offset + j]) / dXbp); // chua tinh muc nuoc trung binh
        f3[i] = dT * VISCOIDX[i * offset + j] / dXbp - dTchia2dX * u[i * offset + j];
    

        if (H_moi[i * offset + j - 1] <= H_TINH){
            if (vtb < 0){
                p = vtb * (-3 * u[i * offset + j] + 4 * u[i * offset + j + 1] - u[i * offset + j + 2]) / dY2;
                q = (u[i * offset + j] - 2 * u[i * offset + j + 1] + u[i * offset + j + 2]) / dYbp;
            }
        }
        else{
            if (H_moi[i * offset + j + 1] <= H_TINH){
                if ((H_moi[i * offset + j - 2] > H_TINH) && (vtb > 0)){
                    p = vtb * (3 * u[i * offset + j] - 4 * u[i * offset + j - 1] + u[i * offset + j - 2]) / dY2;
                    q = (u[i * offset + j] - 2 * u[i * offset + j - 1] + u[i * offset + j - 2] ) / dYbp;
                }
            }else{
                p = vtb * (u[i * offset + j + 1] - u[i * offset + j - 1]) / dY2;
                q = (u[i * offset + j + 1] - 2 * u[i * offset + j] + u[i * offset + j - 1]) / dYbp;
            }
        }

        f5[i] = -2 * u[i * offset + j] + dT * p  - dT * CORIOLIS_FORCE * vtb - dT * VISCOIDX[i * offset + j] * q - dT * (Windx() - Tsxw[i * offset + j]) / Htdu[i * offset + j];
    }

    for (int i = first; i <= last; i++){
        f6[i] = dTchia2dX * Htdu[i * offset + j];
        f7[i] = - dTchia2dX * Htdu[(i - 1) * offset + j];
        //printf("Htdu[%d, %d]: %f\n", i - 1, j, Htdu[(i - 1) * offset + j]);
        f8[i] = z[i * offset + j] - dTchia2dY * (Htdv[i * offset + j] * v[i * offset + j] - Htdv[i * offset + j - 1] * v[i * offset + j - 1]);
    
        a2[i] = f7[i];
        c2[i] = f6[i];
        d2[i] = f8[i];
    }

    update_abcd_at_l(first, last, bienran1, bienran2, a1, b1, c1, d1, f1, f2, f3, f4, f5, f6, f7, f8);


    int sn = boundary_config(true, first, last, bienran1, bienran2, 
            ubp[j], ubt[j], t_z[last * offset + j], t_z[first * offset + j], bienQ[2], bienQ[3], N, a1, b1, c1, d1, a2, b2, c2, d2, AA, BB, CC, DD);

    if (sn > 0)
        tridiag(sn, AA, BB, CC, DD, x, Ap, Bp, ep);

    if (bienran1){
        for (int i = first; i < last; i++) {
            t_z[i * offset + j] = x[2 * (i - first) + 1];
            t_u[i * offset + j] = x[2 * (i - first) + 2];
        }
        t_u[(first - 1) * offset + j] = 0;
    }else{
        if ((bienQ[2]) && (first == 2)){
            for (int i = first; i < last; i++){
                t_z[i * offset + j] = x[2 * (i - first) + 1];
                t_u[i * offset + j] = x[2 * (i - first) + 2];
            }
            t_u[(first - 1) * offset + j] = ubt[j];
        }
        else{
       
            t_u[first * offset + j] = x[1];
            t_u[(first - 1) * offset + j] = (d2[first] - t_z[first * offset + j] - c2[first] * t_u[first * offset + j]) / a2[first];
            for (int i  = first + 1; i < last; i ++){
                t_z[i * offset + j] = x[2 * (i - first)];
                t_u[i * offset + j] = x[2 * (i - first) + 1];
            }
        }
    }       

    if (bienran2){
        t_u[last * offset + j] = 0;
        t_z[last * offset + j] = x[sn];
    }
    else{
        if ((bienQ[3]) && (last == N)){
            t_u[last * offset + j] = ubp[j];
            t_z[last * offset + j] = x[sn];
        }
        else
            //print "long z2"
            t_u[last * offset + j] = (d2[last] - a2[last] * t_u[(last - 1) * offset + j] - t_z[last * offset + j]) / c2[last];
    }
}

__device__ void vzSolver(int offset, int N, int M, int first, int last, int ipos, bool bienran1, bool bienran2, 
    DOUBLE dXbp, DOUBLE dYbp, DOUBLE dT, DOUBLE dTchia2dX, DOUBLE dTchia2dY, DOUBLE* Tsyw, // constants
    DOUBLE* v, DOUBLE* u, DOUBLE* z, DOUBLE* Htdu, DOUBLE* Htdv, DOUBLE* VISCOIDX, DOUBLE* t_v, DOUBLE* t_z, 
    DOUBLE* vbt, DOUBLE* vbd, DOUBLE* H_moi, DOUBLE* Ky1, bool* bienQ,
    DOUBLE *f1, DOUBLE *f2, DOUBLE *f3, DOUBLE *f5, DOUBLE *f6, DOUBLE *f7, DOUBLE *f8,
    DOUBLE *a1, DOUBLE* b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2,
    DOUBLE *AA, DOUBLE *BB, DOUBLE *CC, DOUBLE *DD, DOUBLE *Ap, DOUBLE *Bp, DOUBLE* ep, DOUBLE* x){
    int i = ipos;
    DOUBLE f4 = 2 * g * dTchia2dY;
    DOUBLE p = 0.0;
    DOUBLE q = 0.0;

  

    for (int j = first; j < last; j++){
        DOUBLE utb = (u[(i - 1) * offset + j] + u[i * offset + j] + u[(i - 1) * offset + j + 1] + u[i * offset + j + 1]) * 0.25;
        f1[j] = dTchia2dY * v[i * offset + j] + VISCOIDX[i * offset + j] * dT / dYbp;
        f2[j] = -(2 + Ky1[i * offset + j] * dT * sqrt(v[i * offset + j] * v[i * offset + j] + utb * utb) / Htdv[i * offset + j] + (2 * dT * VISCOIDX[i * offset + j]) / dYbp);
        f3[j] = dT * VISCOIDX[i * offset + j] / dYbp - dTchia2dY * v[i * offset + j];

        if (H_moi[(i - 1) * offset + j] <= H_TINH){
            if (utb < 0){
                q = utb * (-3 * v[i * offset + j] + 4 * v[(i + 1) * offset + j] - v[(i + 2) * offset + j]) / dX2;
                p = (v[i * offset + j] - 2 * v[(i + 1) * offset + j] + v[(i + 2) * offset + j] ) / dXbp;
            }
        }else{
            if (H_moi[(i + 1) * offset + j] <= H_TINH){
                if ((H_moi[(i - 2) * offset + j] > H_TINH) && (utb > 0)){
                    q = utb * (3 * v[i * offset + j] - 4 * v[(i - 1) * offset + j] + v[(i - 2) * offset + j]) / dX2;
                    p = (v[i * offset + j] - 2 * v[(i - 1) * offset + j] + v[(i - 2) * offset + j] ) / dXbp;
                }
            }else{
                q = utb * (v[(i + 1) * offset + j] - v[(i - 1) * offset + j]) / dX2;
                p = (v[(i + 1) * offset + j] - 2 * v[i * offset + j] + v[(i - 1) * offset + j]) / dXbp;
            }
        }
        f5[j] = -2 * v[i * offset + j] + dT * q + dT * CORIOLIS_FORCE * utb - dT * VISCOIDX[i * offset + j] * p - dT * (Windy() - Tsyw[i * offset + j]) / Htdv[i * offset + j];
    }


    for (int j = first; j <= last; j++){
        f6[j] = dTchia2dY * Htdv[i * offset + j];             
        f7[j] = - dTchia2dY * Htdv[i * offset + j - 1];
        f8[j] = z[i * offset + j] - dTchia2dX * (Htdu[i * offset + j] * u[i * offset + j] - Htdu[(i - 1) * offset + j] * u[(i - 1) * offset + j]);
        //printf("gpu: z %f, htdu: %f, u: %f \n", z[i * offset + j], Htdu[i * offset + j], u[i * offset + j] );
        a2[j] = f7[j];
        c2[j] = f6[j];
        d2[j] = f8[j];
    }
   
    update_abcd_at_l(first, last, bienran1, bienran2, a1, b1, c1, d1, f1, f2, f3, f4, f5, f6, f7, f8);

    int sn = boundary_config(false, first, last, bienran1, bienran2, 
        vbt[i], vbd[i], t_z[i * offset + last], t_z[i * offset + first], bienQ[1], bienQ[0], M, a1, b1, c1, d1, a2, b2, c2, d2, AA, BB, CC, DD);
    if (sn > 0)
        tridiag(sn, AA, BB, CC, DD, x, Ap, Bp, ep);

    if (bienran1){
        for (int j = first; j < last; j++){
            t_z[i * offset + j] = x[2 * (j - first) + 1];
            t_v[i * offset + j] = x[2 * (j - first) + 2];
        }
        t_v[i * offset + first - 1] = 0;
    }
    else{
        if( (bienQ[1]) && (first == 2)){
            for (int j = first; j < last; j++){
                t_z[i * offset + j] = x[2 * (j - first) + 1];
                t_v[i * offset + j] = x[2 * (j - first) + 2];
            }
            t_v[i * offset + first - 1] = vbd[i];
        }
        else{
            t_v[i * offset + first] = x[1];
            t_v[i * offset + first - 1] = (d2[first] - t_z[i * offset + first] - c2[first] * t_v[i * offset + first]) / a2[first];
            for (int j = first + 1; j < last; j++){
                t_z[i * offset + j] = x[2 * (j - first)];
                t_v[i * offset + j] = x[2 * (j - first) + 1];
            }
        }
    }

    if (bienran2){
        t_v[i * offset + last] = 0;
        t_z[i * offset + last] = x[sn];
    }
    else{
        if ((bienQ[0]) && (last == M)){
            t_v[i * offset + last] = vbt[i];
            t_z[i * offset + last] = x[sn];
        }
        else{
            t_v[i * offset + last] = (d2[last] - a2[last] * t_v[i * offset + last - 1] - t_z[i * offset + last]) / c2[last];
        }
    }
}

__device__ void uSolver(int offset, int first, int last, int jpos, bool bienran1, bool bienran2, DOUBLE* VISCOIDX, DOUBLE* Tsxw,
    DOUBLE *v, DOUBLE *t_v, DOUBLE *u, DOUBLE *t_u, DOUBLE *z, DOUBLE *t_z, DOUBLE *Kx1, DOUBLE *Htdu, DOUBLE *H_moi){

    int j = jpos;
    DOUBLE p, q, tmp;
    for (int i = first; i < last; i++){
        p = 0; q = 0; tmp = 0;
        DOUBLE vtb = (v[i * offset + j - 1] + v[i * offset + j] + v[(i + 1) * offset +  j - 1] + v[(i + 1) * offset +  j]) * 0.25;
        DOUBLE t_vtb = (t_v[i * offset + j - 1] + t_v[i * offset + j] + t_v[(i + 1) * offset +  j - 1] + t_v[(i + 1) * offset +  j]) * 0.25;
        p = (u[(i + 1) * offset +  j] - u[(i - 1) * offset +  j]) / dX2;
        p = (HaiChiadT + p + Kx1[i * offset + j] * sqrt(vtb * vtb + u[i * offset + j] * vtb) / Htdu[i * offset + j]);
        //print vtb, ' ', t_vtb
        if (H_moi[i * offset + j - 1] <= H_TINH){
            if (vtb < 0){
                q = t_vtb * (-3 * u[i * offset + j] + 4 * u[i * offset + j + 1] - u[i * offset +  j + 2]) / dY2;
                tmp = (u[i * offset +  j] - 2 * u[i * offset +  j + 1] + u[i * offset +  j + 2] ) / dYbp;
            }
        }
        else
            if (H_moi[i * offset +  j + 1] <= H_TINH)
                    if ((H_moi[i * offset +  j - 2] > H_TINH) && (vtb > 0)){ 
                        q = t_vtb * (3 * u[i * offset +  j] - 4 * u[i * offset +  j - 1] + u[i * offset +  j - 2]) /dY2;
                        tmp = (u[i * offset +  j] - 2 * u[i * offset +  j - 1] + u[i * offset +  j - 2] ) / dYbp;
                    }
            else{
                q = t_vtb * (u[i * offset +  j + 1] - u[i * offset +  j - 1]) / dY2;
                tmp = (u[i * offset +  j + 1] - 2 * u[i * offset +  j] + u[i * offset +  j - 1]) / dYbp;
            }
                //print 'q is calculated in line 395'
        //print q
        q = HaiChiadT * u[i * offset +  j] - q + CORIOLIS_FORCE * t_vtb;
        q = (q - g * (z[(i + 1) * offset +  j] - z[i * offset +  j]) / dX + VISCOIDX[i * offset +  j] * ((u[(i + 1) * offset +  j] - 2 * u[i * offset +  j] + u[(i - 1) * offset +  j]) / dXbp + tmp)) + (Windx() - Tsxw[i * offset +  j]) / Htdu[i * offset +  j];
        //print ' ', dX, ' ', dXbp, ' ', Htdu[i, j]
        t_u[i * offset +  j] = q / p;
    }
    if (bienran1)
        t_u[(first - 1) * offset +  j]  = 0;
    else
        t_u[(first - 1) * offset +  j] = 2 * t_u[first * offset + j] - t_u[(first + 1) * offset + j];
    if (bienran2)
        t_u[last * offset +  j] = 0;
    else
        t_u[last * offset +  j] = 2 * t_u[(last - 1) * offset +  j] - t_u[(last - 2) * offset +  j];
    
}

__device__ void vSolver(int offset, int first, int last, int ipos, bool bienran1, bool bienran2, DOUBLE* VISCOIDX, DOUBLE* Tsyw, 
    DOUBLE *v, DOUBLE *t_v, DOUBLE *u, DOUBLE *t_u, DOUBLE *z, DOUBLE *t_z, DOUBLE *Ky1, DOUBLE *Htdv, DOUBLE *H_moi){

    DOUBLE p, q, tmp;
    int i = ipos;
    for (int j = first; j < last; j++){
        q = 0;
        p = 0;
        tmp = 0;
        DOUBLE utb = (u[(i - 1) * offset +  j] + u[i * offset +  j] + u[(i - 1) * offset +  j + 1] + u[i * offset +  j + 1]) * 0.25;

        DOUBLE t_utb = (t_u[(i - 1) * offset +  j] + t_u[i * offset +  j] + t_u[(i - 1) * offset +  j + 1] + t_u[i * offset +  j + 1]) * 0.25;
        //printf("utb: %.14f, %.14f, %.14f, %d, %d\n", utb, t_utb, q, i, j);
        p = (v[i * offset +  j + 1] - v[i * offset +  j - 1]) / dY2;
        p = (HaiChiadT + p + Ky1[i * offset +  j] * sqrt(utb * utb + v[i * offset +  j] * v[i * offset +  j]) / Htdv[i * offset +  j]);
        //printf("H_moi[%d %d], %.10f \n", i, j, H_moi[i - 1, j]);
        if (H_moi[(i - 1) * offset +  j] <= H_TINH){
           // printf("here484, %d %d\n", i, j);
            if (utb < 0){
                q = t_utb * (-3 * v[i * offset +  j] + 4 * v[(i + 1) * offset +  j] + v[(i + 2) * offset +  j]) / dX2;
                tmp = (v[i * offset +  j] - 2 * v[(i + 1) * offset +  j] + v[(i + 2) * offset +  j] ) / dXbp;
            }
        }
        else{
            //printf("here491, %d %d\n", i, j);
            if (H_moi[(i + 1) * offset +  j] <= H_TINH){
                if ((H_moi[(i - 2) * offset +  j] > H_TINH) && (utb > 0)){
                    //printf("here487, %d %d\n", i, j);
                    q = t_utb * (3 * v[i * offset +  j] - 4 * v[(i - 1) * offset +  j] + v[(i - 2) * offset +  j]) /dX2;
                    tmp = (v[i * offset +  j] - 2 * v[(i - 1) * offset +  j] + v[(i - 2) * offset +  j] ) / dXbp;
                }
            }
            else{
                //printf("here489, %d %d\n", i, j);
                q = t_utb * (v[(i + 1) * offset +  j] - v[(i - 1) * offset +  j]) / dX2;
                tmp = (v[(i + 1) * offset +  j] - 2 * v[i * offset +  j] + v[(i - 1) * offset +  j]) / dXbp;
            }
        }

        //if (i == 4) printf("q1: %.14f %d %d\n", q, i, j);
        q = HaiChiadT * v[i * offset +  j] - q - CORIOLIS_FORCE * t_utb;
        q = (q - g * (z[i * offset +  j + 1] - z[i * offset +  j]) / dY + VISCOIDX[i * offset +  j] * (tmp + (v[i * offset +  j + 1] - 2 * v[i * offset +  j] + v[i * offset +  j - 1]) / dYbp)) + (Windy() - Tsyw[i * offset +  j]) / Htdv[i * offset +  j];
        

        t_v[i * offset +  j] = q / p ;
        //if (i == 4) printf("p: %.14f, q: %.14f, t_v[%d, %d] : %.14f \n", p, q, i, j, t_v[i * offset + j] );

    }

    if (bienran1)
        t_v[i * offset +  first - 1] = 0;
    else{
        t_v[i * offset +  first - 1] = 2 * t_v[i * offset +  first] - t_v[i * offset +  first + 1];
    }
    if (bienran2){
        t_v[i * offset +  last] = 0;
    }
    else{
        t_v[i * offset +  last] = 2 * t_v[i * offset +  last - 1] - t_v[i * offset +  last - 2];
    }
    //printf("t_v[%d, %d] = %f\n",i, j, t_v[i * offset + j] );
    //printf("Htdv[%d, %d] = %f\n",i, j, Htdv[i * offset + j] );
    
}


__device__ void set_boundary_vslice(int N, int M, bool* bienran1, bool* bienran2, int* first, int* last, int index, int k, int h_offset, int* daui, int* cuoii, DOUBLE* h){
    int i = index;
    int offset = 5;
    //printf("mem_offset: %d \n", i * offset + k);
    *first = daui[i * offset + k];
    *last = cuoii[i * offset + k];
    //printf("thread: %d A: dau: %d, cuoi: %d\n", threadIdx.x, *first, *last);

    offset = h_offset;
    if ((*first > 2) || ((*first == 2) && (h[i * offset + *first - 1] + h[(i - 1) * offset + *first - 1]) * 0.5 == NANGDAY))
        *bienran1 = true;
    if ((*last < M) || ((*last == M) && (h[i * offset +  *last] + h[(i - 1) * offset + *last]) * 0.5 == NANGDAY))
        *bienran2 = true;
    //printf("thread: %d B: dau: %d, cuoi: %d\n", threadIdx.x, *first, *last);
}

__device__ void set_boundary_uslice(int N, int M, bool* bienran1, bool* bienran2, int* first, int* last, int index, int k, int h_offset, int* dauj, int* cuoij, DOUBLE* h ){
    int offset = 5;
    int j = index;

    *first = dauj[j * offset +  k];
    *last = cuoij[j * offset + k];

    offset = h_offset;

    DOUBLE *depth = &h[(*first - 1) * offset];
    if ((*first > 2) || ((*first == 2) && (depth[j] + depth[j - 1]) * 0.5 == NANGDAY))
        *bienran1 = true;

    depth = &h[*last * h_offset];
    if ((*last < N) || ((*last == N) && (depth[j] + depth[j - 1]) * 0.5 == NANGDAY))
        *bienran2 = true;
    //printf("thread: %d D: dau: %d, cuoi: %d\n", threadIdx.x, *first, *last);
}


// mem_offset: Memory offset for each 
__global__ void solveUZ(int M, int N, int mem_offset, int startidx, int endidx, int* mocj, int* dauj, int* cuoij,  bool*bienQ, DOUBLE* Tsxw,
    DOUBLE* v, DOUBLE* u, DOUBLE* z, DOUBLE* Htdu, DOUBLE* Htdv, DOUBLE* VISCOIDX, DOUBLE* t_u, DOUBLE* t_z, DOUBLE* h,
    DOUBLE* ubt, DOUBLE* ubp, DOUBLE* H_moi, DOUBLE* Kx1, DOUBLE *f1, DOUBLE *f2, DOUBLE *f3, DOUBLE *f5, DOUBLE *f6, DOUBLE *f7, DOUBLE *f8,
    DOUBLE *a1, DOUBLE* b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2,
    DOUBLE *AA, DOUBLE *BB, DOUBLE *CC, DOUBLE *DD, DOUBLE *Ap, DOUBLE *Bp, DOUBLE* ep, DOUBLE* x ){

        int j = calculate_index(M) + startidx;
        if (j >= endidx) return;
        //if (j == 2) printf("precision check %.16f\n", pow (2.70134219723423422342334134, 2.70134219723423422342334134));
        int row_Thomas = (blockIdx.x*blockDim.x + threadIdx.x) * mem_offset;
        int row_f = (blockIdx.x*blockDim.x + threadIdx.x) * (N + 2);
        //printf("hello from first kernel %d\n", threadIdx.x);
        for (int k = 0; k < mocj[j]; k++){
            // set boundary here
            bool bienran1 = false;
            bool bienran2 = false;
            int first, last;
            int h_offset = M + 3; 
            set_boundary_uslice(N, M, &bienran1, &bienran2, &first, &last, j, k, h_offset, dauj, cuoij, h);
            // debug

            //hipError_t Errtype = hipGetLastError();
            //printf("%s\n",hipGetErrorString(Errtype));


            uzSolver(M + 3, N, M, first, last, j, bienran1, bienran2, dXbp, dYbp, dT, dTchia2dX, dTchia2dY, Tsxw,
            v, u, z, Htdu, Htdv, VISCOIDX, t_u, t_z, ubt, ubp, H_moi, Kx1, bienQ, &f1[row_f], &f2[row_f], &f3[row_f], &f5[row_f], 
            &f6[row_f], &f7[row_f], &f8[row_f], &a1[row_f], &b1[row_f], &c1[row_f], &d1[row_f], &a2[row_f], &b2[row_f], &c2[row_f], &d2[row_f], 
            &AA[row_Thomas], &BB[row_Thomas], &CC[row_Thomas], &DD[row_Thomas], &Ap[row_Thomas], &Bp[row_Thomas], &ep[row_Thomas], &x[row_Thomas]);
        }

    }

__global__ void SolveVZ(int M, int N, int mem_offset,int startidx, int endidx, int* moci, int* daui, int* cuoii,  bool*bienQ, DOUBLE* Tsyw,
    DOUBLE* v, DOUBLE* u, DOUBLE* z, DOUBLE* Htdu, DOUBLE* Htdv, DOUBLE* VISCOIDX, DOUBLE* t_v, DOUBLE* t_z, DOUBLE* h,
    DOUBLE* vbt, DOUBLE* vbd, DOUBLE* H_moi, DOUBLE* Ky1, DOUBLE *f1, DOUBLE *f2, DOUBLE *f3, DOUBLE *f5, DOUBLE *f6, DOUBLE *f7, DOUBLE *f8,
    DOUBLE *a1, DOUBLE* b1, DOUBLE *c1, DOUBLE *d1, DOUBLE *a2, DOUBLE *b2, DOUBLE *c2, DOUBLE *d2,
    DOUBLE *AA, DOUBLE *BB, DOUBLE *CC, DOUBLE *DD, DOUBLE *Ap, DOUBLE *Bp, DOUBLE* ep, DOUBLE* x){
    
    //int i = (blockIdx.x*blockDim.x + threadIdx.x) + startidx;
    int i = calculate_index(M) + startidx;
    if (i >= endidx) return;

    //printf("thread no %d say hello from second kernel\n", blockIdx.x*blockDim.x + threadIdx.x);
    int row_Thomas = (blockIdx.x*blockDim.x + threadIdx.x) * mem_offset;
    int row_f = (blockIdx.x*blockDim.x + threadIdx.x) * (M + 2);
    for (int k = 0; k < moci[i]; k++){
        bool bienran1 = false;
        bool bienran2 = false;
        int first, last;
        int h_offset = M + 3;
        set_boundary_vslice(N, M, &bienran1, &bienran2, &first, &last, i, k, h_offset, daui, cuoii, h);
        vzSolver(M + 3, N, M, first, last, i, bienran1, bienran2, dXbp, dYbp, dT, dTchia2dX, dTchia2dY, Tsyw,
            v, u, z, Htdu, Htdv, VISCOIDX, t_v, t_z, vbt, vbd, H_moi, Ky1, bienQ,&f1[row_f],
            &f2[row_f], &f3[row_f], &f5[row_f], &f6[row_f], &f7[row_f], &f8[row_f], &a1[row_f], &b1[row_f], &c1[row_f], &d1[row_f], &a2[row_f], &b2[row_f], &c2[row_f], &d2[row_f], 
            &AA[row_Thomas], &BB[row_Thomas], &CC[row_Thomas], &DD[row_Thomas], &Ap[row_Thomas], &Bp[row_Thomas], &ep[row_Thomas], &x[row_Thomas] );
    }

}


__global__ void solveU(int N, int M, int startidx, int endidx, DOUBLE* VISCOIDX, DOUBLE* Tsxw, int* mocj, int* dauj, int* cuoij,
    DOUBLE *v, DOUBLE *t_v, DOUBLE *u, DOUBLE *t_u, DOUBLE *z, DOUBLE *t_z, DOUBLE *Kx1, DOUBLE *Htdu, DOUBLE *H_moi, DOUBLE* h){
    int j = calculate_index(M) + startidx;
    if (j >= endidx) return;
    //printf("thread no %d say hello from third kernel\n", blockIdx.x*blockDim.x + threadIdx.x);
        for (int k = 0; k < mocj[j]; k++){
            bool bienran1 = false;
            bool bienran2 = false;
            int first, last;
            int h_offset = M + 3;
            set_boundary_uslice(N, M, &bienran1, &bienran2, &first, &last, j, k, h_offset, dauj, cuoij, h);
            uSolver(M + 3,first, last, j, bienran1, bienran2, VISCOIDX, Tsxw, v, t_v, u, t_u, z, t_z, Kx1, Htdu,H_moi);
    }

    
}


__global__ void solveV(int N, int M, int startidx, int endidx, DOUBLE* VISCOIDX, DOUBLE* Tsyw, int* moci, int* daui, int* cuoii,
    DOUBLE *v, DOUBLE *t_v, DOUBLE *u, DOUBLE *t_u, DOUBLE *z, DOUBLE *t_z, DOUBLE *Ky1, DOUBLE *Htdv, DOUBLE *H_moi, DOUBLE* h){
        int i = calculate_index(M) + startidx;
        if (i >= endidx) return;
        //printf("thread no %d say hello from forth kernel\n", blockIdx.x*blockDim.x + threadIdx.x);
        for (int k = 0; k < moci[i]; k++){
            bool bienran1 = false;
            bool bienran2 = false;
            int first, last;
            int h_offset = M + 3;
            set_boundary_vslice(N, M, &bienran1, &bienran2, &first, &last, i, k, h_offset, daui, cuoii, h);
            vSolver(M +3, first, last, i, bienran1, bienran2, VISCOIDX, Tsyw, v, t_v, u, t_u, z, t_z, Ky1, Htdv, H_moi);
        }

}